#include <cusp/coo_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/print.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/precond/diagonal.h>

#include <cusp/io/matrix_market.h> // save matrix to disk in MatrixMarket format
#include "./include/cusp_device.h"

//hipError_t error; 

void solve_on_device( cusp::coo_matrix<int, float, cusp::host_memory>& coo_host, 
                      cusp::array1d<float, cusp::host_memory>& rhs_host, 
                      cusp::array1d<float, cusp::host_memory>& result_host ) {
    	 															
    // convert COO->HYB on the host and transfer HYB to the device
    //cusp::hyb_matrix<int, float, cusp::device_memory> coo_cusp_device = coo_host;
    
    // transfer COO to the device
    cusp::coo_matrix<int, float, cusp::device_memory> coo_cusp_device = coo_host;
    //cusp::io::write_matrix_market_file( coo_cusp_device, "coo_cusp_device.mtx" );
    //cusp::print(coo_cusp_device);

    // convert COO->DIA on the host and transfer DIA to the device
    //cusp::dia_matrix<int, float, cusp::device_memory> dia_cusp_device = coo_host;
 
    // convert COO->DIA on the device
    //cusp::dia_matrix<int, float, cusp::device_memory> dia_cusp_device;
    //cusp::convert(dia_cusp_device, coo_cusp_device);
	
	 // transfer rhs_host to the device
    cusp::array1d<float, cusp::device_memory> rhs_device = rhs_host;
    //cusp::print(rhs_device);
    //cusp::io::write_matrix_market_file( rhs_device, "rhs_device.mtx" );

	 // transfer result_host to the device	
    cusp::array1d<float, cusp::device_memory> result_device = result_host;
         
    // set stopping criteria (iteration_limit = 2000, relative_tolerance = 1e-7)
    cusp::verbose_monitor<float> monitor(rhs_device, 100, 1e-2);
    
    // set preconditioner (identity) doesn't affect the speed of convergence
    cusp::identity_operator<float, cusp::device_memory> M( coo_cusp_device.num_rows, 
																			  coo_cusp_device.num_rows );

    // solve the linear system A * x = b -> coo_cusp_device * result_device = rhs_device 
    cusp::krylov::cg(coo_cusp_device, result_device, rhs_device, monitor, M);	 
    //cusp::print(x);
    //cusp::io::write_matrix_market_file(result_device, "result_device.mtx");
    
    // return the result on the host 
    // behind each '=' there is a call to cudamalloc
    result_host = result_device; 
    
    //cusp::io::write_matrix_market_file(result_host, "result_host_final.mtx");

}
    // diagonal preconditioner results in NaN
    // cusp::precond::diagonal<float, cusp::device_memory> M( coo_cusp_device ); 
																																					  																		  																		  
