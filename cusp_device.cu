#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>

// uncomment if you want to save matrix to disk in MatrixMarket format
#include <cusp/io/matrix_market.h>
#include "./include/cusp_device.h"

//hipError_t error; 

void solve_on_device( cusp::coo_matrix<int, float, cusp::host_memory>& coo_host, 
                      cusp::array1d<float, cusp::host_memory>& rhs_host, 
                      cusp::array1d<float, cusp::host_memory>& result_host ) {
    	 															
    // transfer COO to the device
    cusp::coo_matrix<int, float, cusp::device_memory> coo_cusp_device = coo_host;
    //cusp::io::write_matrix_market_file( coo_cusp_device, "coo_cusp_device.mtx" );
    //cusp::print(coo_cusp_device);
	
	 // transfer rhs_host to the device
    cusp::array1d<float, cusp::device_memory> rhs_device = rhs_host;
    //cusp::print(rhs_device);
    //cusp::io::write_matrix_market_file( rhs_device, "rhs_device.mtx" );

	 // transfer result_host to the device	
    cusp::array1d<float, cusp::device_memory> result_device = result_host;
         
    // set stopping criteria (iteration_limit = 100, relative_tolerance = 1e-2)
    cusp::verbose_monitor<float> monitor(rhs_device, 100, 1e-2);
    
    // set preconditioner (identity) doesn't affect the speed of convergence
    cusp::identity_operator<float, cusp::device_memory> M( coo_cusp_device.num_rows, 
	 																		  coo_cusp_device.num_rows );

    // solve the linear system A * x = b -> coo_cusp_device * result_device = rhs_device 
    cusp::krylov::cg(coo_cusp_device, result_device, rhs_device, monitor, M);	 
    //cusp::print(x);
    //cusp::io::write_matrix_market_file(result_device, "result_device.mtx");
    
    // return the result on the host 
    // behind each '=' there is a call to cudamalloc
    result_host = result_device; 
    
    //cusp::io::write_matrix_market_file(result_host, "result_host_final.mtx");

}
	// CUSP's preconditioners (diagonal, smoothed_aggregation, approximate inverse) 
	// fail to work. On the linear system, matrices of quaternions are converted 
	// into a system of reals, so it might be just that preconditioners for real 
	// matrices don't work for quaternionic matrices.

    // diagonal preconditioner results in NaN
    // cusp::precond::diagonal<float, cusp::device_memory> M( coo_cusp_device ); 
																																					  																		  																		  
